#include "hip/hip_runtime.h"
/**
 * @file main.cpp
 * @author Munier Louis
 * @date 30.05.19
 * @version 1.0
 *
 * Main file of the nBody project.
 *
 * @mainpage N-Body project
 *
 * Project in the case of the course of Parallel and High Performance Computing at EPFL - Ecole Polytechnique Fédérale
 * de Lausanne.
 *
 * @section Description
 *
 * The n-body problem aims at simulating a dynamical system of particles under the influence of physical forces. We’ll
 * restrain on the gravity field applied on celestial bodies :
 * @f[ F_{ij} = \frac{G m_i m_j (q_j - q_i)}{|| q_j - q_i ||} @f]
 *
 * where @f$ G @f$ is the gravitational constant, @f$ m_i @f$ and @f$ m_j @f$ the masses of the @f$ i @f$-th and
 * @f$j@f$-th bodies and @f$ q_i @f$ and @f$ q_j @f$ their positions.
 *
 * @section Implementation
 *
 * Here the solution is implemented using Brute-force algorithm to have a working parallelization of the algorithm with
 * CUDA and compare with both sequential Barnes-Hut and brute-force without parallel implementation. The user can choose
 * differents parameters as define :
 * - NB_PARTICLES : number of particles
 * - PRINT : save solution in csv file, at each time-step, to display solution animation in programs (e.g. paraview)
 * - DELTA_T : time-step of each iteration
 */

/**
 * @include constants.hpp which contains all the needed project's constants/includes
 */
#include "constants.hpp"

/**
 * __global__ parallelized function to process the random generation of the particle positions. It is done to stress
 * application.
 * 
 * @param rnd values generated for particle positions in x, y and z
 * @param nb_elements to avoid process too much time the same values due to parallelization
 */
__global__ void process_pos(float* rnd, int nb_elements){
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < nb_elements) {
        /** Process random coordinate of particles to have them shifted if they stay in boundaries */
        rnd[i] *= SIDE_2f;
        rnd[i] -= SIDE_4f;

        if (abs(rnd[i] - SHIFT) < SIDE_4f)
            rnd[i] -= SHIFT;
    }
}
 
/**
 * __global__ parallelized function to process the random generation of the mass value. It is done to have value
 * between 0 and MASS_MAX.
 * 
 * @param rnd values generated for particle mass
 * @param nb_elements to avoid process too much time the same values due to parallelization
 */
__global__ void process_mass(float* rnd, int nb_elements){
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < nb_elements) {
        rnd[i] *= MASS_MAX;
    }
}

/**
 * If PRINT is defined, generate a csv file to display animation of the result in external software (e.g. paraview).
 *
 * @param particle pointer on the particle to write in csv file
 * @param iter iteration to change filename
 * @param dir to have filepath where write all the values
 */
#ifdef PRINT
__host__ void generate_file(float* particle, int iter, const std::string& dir) {
    std::ofstream csv_file;
    std::string filename = dir + "/out_" + std::to_string(iter) + ".csv";

    csv_file.open(filename);
    csv_file << "x,y,z\n";

    for (unsigned int i = 0; i < NB_PARTICLES; i++)
        csv_file << particle[DIM_3 * i] << "," << particle[DIM_3 * i + 1] << "," << particle[DIM_3 * i + 2] << std::endl;
    
    csv_file.close();
}
#endif

/**
 * __global__ parallelized function to compute the acceleration for each particle. Each particle have his own thread
 * and compute the load applied on it from each of the others particle.
 *
 * @param pos pointer on the position of each particle in device memory
 * @param acc pointer on the acceleration of each particle in device memory
 * @param mass pointer on the mass of each particle in device memory
 */
__global__ void kernel_compute_acc(float* pos, float* acc, float* mass) {
    float r[DIM_3] = {0.0f};
    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int idx = DIM_3 * i;

    if ( i < NB_PARTICLES ) {
        for (unsigned int j = 0; j < NB_PARTICLES; j++) {
            if (i != j) {
                r[0] = pos[DIM_3 * j] - pos[idx];
                r[1] = pos[DIM_3 * j + 1] - pos[idx + 1];
                r[2] = pos[DIM_3 * j + 2] - pos[idx + 2];

                float distSqrd = sqrtf(r[0] * r[0] + r[1] * r[1] + r[2] * r[2]);
                float s = G * mass[j] / distSqrd;

                acc[idx] += r[0] * s;
                acc[idx + 1] += r[1] * s;
                acc[idx + 2] += r[2] * s;
            }
        }
    }
}

/**
 * __global__ parallelized function to compute the update position and velocity for each particle. Each particle have his
 * own thread and compute the load applied on it from each of the others particle.
 *
 * @param pos pointer on the position of each particle in device memory
 * @param vel pointer on the velocity of each particle in device memory
 * @param acc pointer on the acceleration of each particle in device memory
 * @param mass pointer on the mass of each particle in device memory
 */
__global__ void kernel_update_pos_vel(float* pos, float* vel, float* acc, float* mass) {
    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
    unsigned int idx = DIM_3 * i;

    if (i < NB_PARTICLES) {
        /** Update velocity */
        vel[idx] += acc[idx] * DELTA_T;
        vel[idx + 1] += acc[idx + 1] * DELTA_T;
        vel[idx + 2] += acc[idx + 2] * DELTA_T;

        /** Update position */
        pos[idx] += vel[idx] * DELTA_T;
        pos[idx + 1] += vel[idx + 1] * DELTA_T;
        pos[idx + 2] += vel[idx + 2] * DELTA_T;

        /** Reset acceleration */
        acc[idx] = 0.0f;
        acc[idx + 1] = 0.0f;
        acc[idx + 2] = 0.0f;
    }
}

/**
 * __host__ function that call both comput_acc and update_pos_vel functions. It Allocate also the number of block needed
 * to compute all the values. A synchronization function is also called to avoid having unfinished computation on a block
 * before calling the second function.
 *
 * @param pos pointer on the position of each particle in device memory
 * @param vel pointer on the velocity of each particle in device memory
 * @param acc pointer on the acceleration of each particle in device memory
 * @param mass pointer on the mass of each particle in device memory
 */
__host__ void update_particles(float* pos, float* vel, float* acc, float* mass) {
    int threadsPerBlock = min(MAX_THREADS, DIM_3 * NB_PARTICLES);
    int blocksPerGrid = (DIM_3 * NB_PARTICLES + threadsPerBlock - 1) / threadsPerBlock;
    kernel_compute_acc<<< blocksPerGrid, threadsPerBlock >>>(pos, acc, mass);

    hipDeviceSynchronize();
    kernel_update_pos_vel<<< blocksPerGrid, threadsPerBlock >>>(pos, vel, acc, mass);
}

/**
 * Main function, compute time to solve problem and store size of the overall area where particle are studied.
 * In this function, all the memory is also allocated on the host and device part.
 *
 * @param argc default input in c++ main function
 * @param argv default input in c++ main function
 * @return success if no errors are reached
 */
int main(int argc, char *argv[]) {
    // Utilities to track the time
    hipEvent_t start;
    hipEvent_t stop;
    float msecTotal(0.0f);

    // Filename to store particles
    std::string dir("output");

    // allocate host memory for pos, vel and mass
    unsigned int mem_size_pos = sizeof(float) * DIM_3 * NB_PARTICLES;
    float* h_pos = (float*) malloc(mem_size_pos);
    
    unsigned int mem_size_vel = sizeof(float) * DIM_3 * NB_PARTICLES;
    float* h_vel = (float*) malloc(mem_size_vel);
    
    unsigned int mem_size_mass = sizeof(float) * NB_PARTICLES;
    float* h_mass = (float*) malloc(mem_size_mass);
    //float flop = (float) * (NB_PARTICLES * NB_PARTICLES + NB_PARTICLES) * DIM_3;
    
    // allocate device memory
    float* d_pos;
    hipMalloc((void**) &d_pos, mem_size_pos);
    hipMemset(d_pos, 0.0f, mem_size_pos);

    float* d_vel;
    hipMalloc((void**) &d_vel, mem_size_vel);
    hipMemset(d_vel, 0.0f, mem_size_vel);

    float* d_mass;
    hipMalloc((void**) &d_mass, mem_size_mass);
    hipMemset(d_mass, 0.0f, mem_size_mass);

    // allocate device memory for result
    unsigned int mem_size_acc = sizeof(float) * DIM_3 * NB_PARTICLES;
    float* d_acc;
    hipMalloc((void**) &d_acc, mem_size_acc);

    // allocate host memory for the result
    float* h_acc = (float*) malloc(mem_size_acc);

    // create and start timer
    hipEventCreate(&start);
    hipEventRecord(start, NULL);

    // copy host memory to device
    hipMemcpy(d_pos, h_pos, mem_size_pos, hipMemcpyHostToDevice);
    hipMemcpy(d_vel, h_vel, mem_size_vel, hipMemcpyHostToDevice);

    // Generate random numbers on device
    hiprandGenerator_t gen;

    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);                 /* Create pseudo-random number generator */
    hiprandSetPseudoRandomGeneratorSeed(gen, (unsigned long long) start);    /* Set seed */
    hiprandGenerateUniform(gen, d_pos, DIM_3 * NB_PARTICLES);                /* Generate floats pos on device */
    hiprandGenerateUniform(gen, d_mass, NB_PARTICLES);                       /* Generate floats mass on device */

    // setup execution parameters
    int threadsPerBlock = min(MAX_THREADS, DIM_3 * NB_PARTICLES);
    int blocksPerGrid = (DIM_3 * NB_PARTICLES + threadsPerBlock - 1) / threadsPerBlock;
    process_pos<<< blocksPerGrid, threadsPerBlock >>>(d_pos, DIM_3 * NB_PARTICLES);

    blocksPerGrid = (NB_PARTICLES + threadsPerBlock - 1) / threadsPerBlock;
    process_mass<<< blocksPerGrid, threadsPerBlock >>>(d_mass, NB_PARTICLES);

    for (unsigned int k = 0; k < ITERATIONS; k++) {
        update_particles(d_pos, d_vel, d_acc, d_mass);
        hipDeviceSynchronize();

#ifdef PRINT
        // copy result from device to host
        hipMemcpy(h_pos, d_pos, mem_size_pos, hipMemcpyDeviceToHost);
        generate_file(h_pos, k, dir);
#endif
    }

    hipMemcpy(h_pos, d_pos, mem_size_pos, hipMemcpyDeviceToHost);
    hipMemcpy(h_vel, d_vel, mem_size_vel, hipMemcpyDeviceToHost);
    hipMemcpy(h_mass, d_mass, mem_size_mass, hipMemcpyDeviceToHost);
    hipMemcpy(h_acc, d_acc, mem_size_acc, hipMemcpyDeviceToHost);

    // stop and destroy timer
    hipEventCreate(&stop);
    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&msecTotal, start, stop);

    //printf("Processing time: %f (ms), GFLOPS: %f \n", msecTotal, flop / msecTotal/ 1e+6);
    printf("Processing time: %f (ms)\n", msecTotal);

    /** Print all the parameters */
    std::cout << "-- Brut force parallelized --" << std::endl;
    std::cout << "Epsilon " << EPSILON << std::endl;
    std::cout << "Nb particles " << NB_PARTICLES << std::endl;
    std::cout << "Nb dimensions " << DIM_3 << std::endl;
    std::cout << "Side " << SIDE << std::endl;
    std::cout << "Shift " << SHIFT << std::endl;
    std::cout << "Occupation percentage " << OCCUPATION_PERC << std::endl;
    std::cout << "Maximum mass " << MASS_MAX << std::endl;
    std::cout << "Delta t " << DELTA_T << std::endl;
    std::cout << "Nb iterations " << ITERATIONS << std::endl;

    // clean up memory
    free(h_pos);
    free(h_vel);
    free(h_acc);
    free(h_mass);
    
    hipFree(d_pos);
    hipFree(d_vel);
    hipFree(d_acc);
    hipFree(d_mass);

    hiprandDestroyGenerator(gen);
    exit(EXIT_SUCCESS);
}