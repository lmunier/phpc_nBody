#include "hip/hip_runtime.h"
/**
 * @file main.cpp
 * @author Munier Louis
 * @date 30.05.19
 * @version 1.0
 *
 * Main file of the nBody project.
 *
 * @mainpage N-Body project
 *
 * Project in the case of the course of Parallel and High Performance Computing at EPFL - Ecole Polytechnique Fédérale
 * de Lausanne.
 *
 * @section Description
 *
 * The n-body problem aims at simulating a dynamical system of particles under the influence of physical forces. We’ll
 * restrain on the gravity field applied on celestial bodies :
 * @f[ F_{ij} = \frac{G m_i m_j (q_j - q_i)}{|| q_j - q_i ||} @f]
 *
 * where @f$ G @f$ is the gravitational constant, @f$ m_i @f$ and @f$ m_j @f$ the masses of the @f$ i @f$-th and
 * @f$j@f$-th bodies and @f$ q_i @f$ and @f$ q_j @f$ their positions.
 *
 * @section Implementation
 *
 * The solution is implemented using Barnes-Hut algorithm and quadtree/octree data-structure. The particularity is that
 * the user can choose differents parameters as define :
 * - NB_DIM : set at DIM_2 or DIM_3 to choose between 2D and 3D problem solution
 * - NB_PARTICLES : number of particles
 * - PRINT : save solution in csv file, at each time-step, to display solution animation in programs (e.g. paraview)
 * - DELTA_T : time-step of each iteration
 */

/**
 * @include constants.hpp which contains all the needed project's constants/includes
 * @include Vector.hpp custom library to have minimal vector implementation
 * @include Tree.hpp library to create a quadtree/octree data structure and interact on different cells/particles
 */
#include "constants.hpp"
#include "Particle.hpp"

#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)

#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)
  
__global__ void process_pos(float* rnd, int nb_elements){
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < nb_elements) {
        /** Generate random coordinate for a new particle and shift it if it stays in boundaries
         * to stress application */
        rnd[i] *= SIDE_2f;
        rnd[i] -= SIDE_4f;

        if (abs(rnd[i] - SHIFT) > SIDE_4f)
            rnd[i] -= SHIFT;
    }
}
  
__global__ void process_mass(float* rnd, int nb_elements){
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < nb_elements) {
        /** Generate random coordinate for a new particle and shift it if it stays in boundaries
         * to stress application */
        rnd[i] *= MASS_MAX;
    }
}

/**
 * Update the load applied to a particle by implementation a Depth First Search on the quadtree/octree data-structure.
 *
 * @tparam Type of the vector, 2D or 3D (and int, float, etc ...)
 * @param head pointer to the current cell of the tree
 * @param part_loaded pointer to the current particle for which the load is computed
 */
// void update_load(thrust::device_vector< Particle<Type>* >* particles) {
//     for (auto p_i : *particles) {
//         Type load = Type();

//         for (auto p_j : *particles) {
//             Type tmp = (*p_j).get(POS) - (*p_i).get(POS);
//             float d = max(tmp.norm(), EPSILON);
//             load = load + tmp * (G * (*p_i).get_mass() * (*p_j).get_mass()) / d;
//         }

//         (*p_i).set(LOAD, load);
//     }
// }

/**
 * Update position and velocity for each particle. Generate a csv file with all the position if needed.
 *
 * @tparam Type of the vector, 2D or 3D (and int, float, etc ...)
 * @param root pointer on the node of the previous cell
 * @param iter current iteration of the solution
 */
// void update_particles_pos(vector< Particle<Type>* >* particles, Type dim, int iter, const string& dir){
//     for (auto p : *particles) {
//         p->update_vel_pos();

//         if (p->is_out_boundaries(dim))
//             delete p;

// #ifdef PRINT
//         generate_file(p, 1000 * iter * DELTA_T, dir);
// #endif
//     }
// }

/**
 * If PRINT is defined, generate a csv file to display animation of the result in external software (e.g. paraview).
 *
 * @tparam Type of the vector, 2D or 3D (and int, float, etc ...)
 * @param particle pointer on the particle to write in csv file
 * @param millis_time timestep to change filename and save chronology
 */
// #ifdef PRINT
// void generate_file(AbstractType<Type>* particle, int millis_time, const string& dir) {
//     ofstream csv_file;
//     string filename = dir + "/out_" + to_string(millis_time) + ".csv";

//     csv_file.open(filename, ios::app);

//     /** Check if file is empty to write the title of each column */
//     if (csv_file.tellp() == 0) {
// #if NB_DIM == DIM_2
//         csv_file << "x,y\n";
// #elif NB_DIM == DIM_3
//         csv_file << "x,y,z\n";
// #endif
//     }

//     csv_file << particle->get(POS).to_file();
//     csv_file.close();
// }
// #endif

/**
 * Main function, compute time to solve problem and store size of the overall area where particle are studied.
 *
 * @param argc default input in c++ main function
 * @param argv default input in c++ main function
 * @return success if no errors are reached
 */
int main(int argc, char *argv[]) {
    // Utilities to track the time
    hipEvent_t start;
    hipEvent_t stop;
    float msecTotal(0.0f);
    
    float part_mass[NB_PARTICLES] = {0.0f};
    float part_pos[NB_DIM * NB_PARTICLES] = {0.0f};
    float part_vel[NB_DIM * NB_PARTICLES] = {0.0f};
    float part_load[NB_DIM * NB_PARTICLES] = {0.0f};

    // Filename to store particles
    std::string dir("");

    if (argv[1])
        dir = argv[1];
    else
        dir = "../output";

    // allocate host memory for pos, vel and load
    unsigned int mem_size_pos = sizeof(part_pos);
    float* h_pos = (float*) malloc(mem_size_pos);
    
    unsigned int mem_size_vel = sizeof(part_vel);
    float* h_vel = (float*) malloc(mem_size_vel);
    
    unsigned int mem_size_mass = sizeof(part_mass);
    float* h_mass = (float*) malloc(mem_size_mass);
    //float flop = 2 * (float)WC * (float)HC * (float)WA;
    
    // allocate device memory
    float* d_pos;
    hipMalloc((void**) &d_pos, mem_size_pos);
    hipMemset(d_pos, 0.0f, mem_size_pos);

    float* d_vel;
    hipMalloc((void**) &d_vel, mem_size_vel);
    hipMemset(d_vel, 0.0f, mem_size_vel);

    float* d_mass;
    hipMalloc((void**) &d_mass, mem_size_mass);
    hipMemset(d_mass, 0.0f, mem_size_mass);

    // allocate device memory for result
    unsigned int mem_size_load = sizeof(part_load);
    float* d_load;
    hipMalloc((void**) &d_load, mem_size_load);

    // allocate host memory for the result
    float* h_load = (float*) malloc(mem_size_load);

    dim3 threads, grid;

    // create and start timer
    hipEventCreate(&start);
    hipEventRecord(start, NULL);

    // copy host memory to device
    hipMemcpy(d_pos, h_pos, mem_size_pos, hipMemcpyHostToDevice);
    hipMemcpy(d_vel, h_vel, mem_size_vel, hipMemcpyHostToDevice);

    // Generate random numbers on device
    hiprandGenerator_t gen;

    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);     /* Create pseudo-random number generator */
    hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);           /* Set seed */
    hiprandGenerateUniform(gen, d_pos, NB_DIM * NB_PARTICLES);   /* Generate floats pos on device */
    hiprandGenerateUniform(gen, d_mass, NB_PARTICLES);   /* Generate floats mass on device */

    // setup execution parameters
    int threadsPerBlock = THREADS;
    int blocksPerGrid = (NB_DIM * NB_PARTICLES + threadsPerBlock - 1) / threadsPerBlock;
    process_pos<<< blocksPerGrid, threadsPerBlock >>>(d_pos, NB_DIM * NB_PARTICLES);

    blocksPerGrid = (NB_PARTICLES + threadsPerBlock - 1) / threadsPerBlock;
    process_mass<<< blocksPerGrid, threadsPerBlock >>>(d_mass, NB_PARTICLES);

    // copy result from device to host
    hipMemcpy(h_pos, d_pos, mem_size_pos, hipMemcpyDeviceToHost);
    hipMemcpy(h_vel, d_vel, mem_size_vel, hipMemcpyDeviceToHost);
    hipMemcpy(h_mass, d_mass, mem_size_mass, hipMemcpyDeviceToHost);
    hipMemcpy(h_load, d_load, mem_size_load, hipMemcpyDeviceToHost);

    // stop and destroy timer
    hipEventCreate(&stop);
    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&msecTotal, start, stop);
    //printf("Processing time: %f (ms), GFLOPS: %f \n", msecTotal, flop / msecTotal/ 1e+6);
    printf("Processing time: %f (ms)\n", msecTotal);

    /** Print all the parameters */
    std::cout << "Brut force" << std::endl;
    std::cout << "Epsilon " << EPSILON << std::endl;
    std::cout << "Nb particles " << NB_PARTICLES << std::endl;
    std::cout << "Nb dimensions " << NB_DIM << std::endl;
    std::cout << "Side " << SIDE << std::endl;
    std::cout << "Shift " << SHIFT << std::endl;
    std::cout << "Occupation percentage " << OCCUPATION_PERC << std::endl;
    std::cout << "Maximum mass " << MASS_MAX << std::endl;
    std::cout << "Delta t " << DELTA_T << std::endl;
    std::cout << "Nb iterations " << ITERATIONS << std::endl;

    // /* Show result */
    // for(unsigned int i = 0; i < NB_PARTICLES; i++) {
    //     printf("%1.4f %1.4f %1.4f \t %1.4f \n", h_pos[i * NB_DIM], h_pos[i * NB_DIM + 1], h_pos[i * NB_DIM + 2], h_mass[i]);
    // }

    //std::cout << exec_time << std::endl;

    // clean up memory
    free(h_pos);
    free(h_vel);
    free(h_load);
    free(h_mass);
    
    hipFree(d_pos);
    hipFree(d_vel);
    hipFree(d_load);
    hipFree(d_mass);

    hiprandDestroyGenerator(gen);
    exit(EXIT_SUCCESS);

    return 0;
}